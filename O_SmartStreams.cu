#include "hip/hip_runtime.h"
// Name:Tanner Wells 
// Two streams overlapped smartly 
// nvcc 15SmartStreams.cu -o temp
/*
 What to do:
 Read about CUDA stream cooperation.

This code provides most of the setup needed to create two CUDA streams. 
Complete the implementation by replacing all the ???s.

Once the two streams are working, overlap them in a smart way to improve performance.
*/

/*
 Purpose:
 To learn how to use CUDA streams intelligently.
*/

// Include files
#include <sys/time.h>
#include <stdio.h>

// Defines
#define DATA_CHUNKS (1024*1024) 
#define ENTIRE_DATA_SET (20*DATA_CHUNKS)
#define MAX_RANDOM_NUMBER 1000
#define BLOCK_SIZE 256

//Globals
dim3 BlockSize; //This variable will hold the Dimensions of your block
dim3 GridSize; //This variable will hold the Dimensions of your grid
float *NumbersOnGPU, *PageableNumbersOnCPU, *PageLockedNumbersOnCPU;
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A0_GPU, *B0_GPU, *C0_GPU, *A1_GPU, *B1_GPU, *C1_GPU; //GPU pointers
hipEvent_t StartEvent, StopEvent;
hipStream_t Stream0, Stream1;

//Function prototypes
void cudaErrorCheck(const char *, int);
void setUpCudaDevices();
void allocateMemory();
void loadData();
void cleanUp();
__global__ void trigAdditionGPU(float *, float *, float *, int );

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

//This will be the layout of the parallel space we will be using.
void setUpCudaDevices()
{
	hipEventCreate(&StartEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventCreate(&StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipDeviceProp_t prop;
	int whichDevice;
	
	hipGetDevice(&whichDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipGetDeviceProperties(&prop, whichDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	
	if(prop.deviceOverlap != 1)
	{
		printf("\n GPU will not handle overlaps so no speedup from streams");
		printf("\n Good bye.");
		exit(0);
	}
	
	hipStreamCreate(&Stream0);
	cudaErrorCheck(__FILE__, __LINE__);
	hipStreamCreate(&Stream1);
	cudaErrorCheck(__FILE__, __LINE__);
	
	BlockSize.x = BLOCK_SIZE;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	if(DATA_CHUNKS%BLOCK_SIZE != 0)
	{
		printf("\n Data chunks do not divide evenly by block size, sooo this program will not work.");
		printf("\n Good bye.");
		exit(0);
	}
	GridSize.x = DATA_CHUNKS/BLOCK_SIZE;
	GridSize.y = 1;
	GridSize.z = 1;	
}

//Sets a side memory on the GPU and CPU for our use.
void allocateMemory()
{	
	//Allocate Device (GPU) Memory
	hipMalloc(&A0_GPU,DATA_CHUNKS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B0_GPU,DATA_CHUNKS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C0_GPU,DATA_CHUNKS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&A1_GPU,DATA_CHUNKS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B1_GPU,DATA_CHUNKS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C1_GPU,DATA_CHUNKS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	
	//Allocate page locked Host (CPU) Memory
	hipHostAlloc(&A_CPU, ENTIRE_DATA_SET*sizeof(float), hipHostMallocDefault);
	cudaErrorCheck(__FILE__, __LINE__);
	hipHostAlloc(&B_CPU, ENTIRE_DATA_SET*sizeof(float), hipHostMallocDefault);
	cudaErrorCheck(__FILE__, __LINE__);
	hipHostAlloc(&C_CPU, ENTIRE_DATA_SET*sizeof(float), hipHostMallocDefault);
	cudaErrorCheck(__FILE__, __LINE__);
}

void loadData()
{
	time_t t;
	srand((unsigned) time(&t));
	
	for(int i = 0; i < ENTIRE_DATA_SET; i++)
	{		
		A_CPU[i] = MAX_RANDOM_NUMBER*rand()/RAND_MAX;
		B_CPU[i] = MAX_RANDOM_NUMBER*rand()/RAND_MAX;	
	}
}

//Cleaning up memory after we are finished.
void cleanUp()
{
	hipFree(A0_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(B0_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(C0_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(A1_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(B1_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(C1_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipHostFree(A_CPU);
	cudaErrorCheck(__FILE__, __LINE__);
	hipHostFree(B_CPU);
	cudaErrorCheck(__FILE__, __LINE__);
	hipHostFree(C_CPU);
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipEventDestroy(StartEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventDestroy(StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipStreamDestroy(Stream0);
	cudaErrorCheck(__FILE__, __LINE__);
	hipStreamDestroy(Stream1);
	cudaErrorCheck(__FILE__, __LINE__);
}

__global__ void trigAdditionGPU(float *a, float *b, float *c, int n)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(id < n)
	{
		c[id] = sin(a[id]) + cos(b[id]);
	}
}

int main()
{
	float timeEvent;
	
	setUpCudaDevices();
	allocateMemory();
	loadData();
	
	hipEventRecord(StartEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);

	/*These streams work the same as the prevoius single stream. However now each stream can work independtly on each chunk of the input.
	In stream 0 the code queues asynchronous copies of a and b to the GPU then it queues trigAdditionGPU kernel, and then queues a copy back to c. 
	The second stream does the same exact thing with the next chunk of data but through stream 1. This for loop runs alternating each stream until 
	it has queued every peice of input data waiting to be processed.*/
	for(int i = 0; i < ENTIRE_DATA_SET; i += DATA_CHUNKS*2)
	{
		//This is going to be the first stream (stream 0)

		hipMemcpyAsync(A0_GPU, A_CPU + i, DATA_CHUNKS * sizeof(float), hipMemcpyHostToDevice, Stream0);
		cudaErrorCheck(__FILE__, __LINE__);
		hipMemcpyAsync(B0_GPU, B_CPU + i, DATA_CHUNKS * sizeof(float), hipMemcpyHostToDevice, Stream0);
		cudaErrorCheck(__FILE__, __LINE__);

		trigAdditionGPU<<<GridSize, BlockSize, 0, Stream0>>>(A0_GPU, B0_GPU, C0_GPU, DATA_CHUNKS);

		hipMemcpyAsync(C_CPU + i, C0_GPU, DATA_CHUNKS * sizeof(float), hipMemcpyDeviceToHost, Stream0);
		
		//This next is going to be the same as stream 0 however this is going to be our second stream (stream 1).
		hipMemcpyAsync(A1_GPU, A_CPU + i + DATA_CHUNKS, DATA_CHUNKS * sizeof(float), hipMemcpyHostToDevice, Stream1);
		cudaErrorCheck(__FILE__, __LINE__);
		hipMemcpyAsync(B1_GPU, B_CPU + i + DATA_CHUNKS, DATA_CHUNKS * sizeof(float), hipMemcpyHostToDevice, Stream1);
		cudaErrorCheck(__FILE__, __LINE__);

		trigAdditionGPU<<<GridSize, BlockSize, 0, Stream1>>>(A1_GPU, B1_GPU, C1_GPU, DATA_CHUNKS);

		hipMemcpyAsync(C_CPU + i + DATA_CHUNKS, C1_GPU, DATA_CHUNKS * sizeof(float), hipMemcpyDeviceToHost, Stream1);
		cudaErrorCheck(__FILE__, __LINE__);
	}
	
	// Make the CPU wait until the Streams have finishd before it continues.
	hipStreamSynchronize(Stream0);
	hipStreamSynchronize(Stream1);
	
	hipEventRecord(StopEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	// Make the CPU wiat until this event finishes so the timing will be correct.
	hipEventSynchronize(StopEvent); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU = %3.1f milliseconds", timeEvent);
	
	
	printf("\n");
	//You're done so cleanup your mess.
	cleanUp();	
	
	return(0);
}
