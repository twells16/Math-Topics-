// Name: Tanner Wells
// Vector addition on the GPU of any size with fixed block and grid size also adding pragma unroll for speed up.
// nvcc HW4.cu -o temp
/*
 What to do:
 This is the solution to HW3. It works well for adding vectors with fixed-size blocks. 
 Given the size of the vector it needs to add, it takes a set block size, determines how 
 many blocks are needed, and creates a grid large enough to complete the task. Cool, cool!
 
 But—and this is a big but—this can get you into trouble because there is a limited number 
 of blocks you can use. Though large, it is still finite. Therefore, we need to write the 
 code in such a way that we don't have to worry about this limit. Additionally, some block 
 and grid sizes work better than others, which we will explore when we look at the 
 streaming multiprocessors.
 
 Extend this code so that, given a block size and a grid size, it can handle any vector addition. 
 Start by hard-coding the block size to 256 and the grid size to 64. Then, experiment with different 
 block and grid sizes to see if you can achieve any speedup. Set the vector size to a very large value 
 for time testing.

 You’ve probably already noticed that the GPU doesn’t significantly outperform the CPU. This is because 
 we’re not asking the GPU to do much work, and the overhead of setting up the GPU eliminates much of the 
 potential speedup. 
 
 To address this, modify the computation so that:
 c = sqrt(cos(a)*cos(a) + a*a + sin(a)*sin(a) - 1.0) + sqrt(cos(b)*cos(b) + b*b + sin(b)*sin(b) - 1.0)
 Hopefully, this is just a convoluted and computationally expensive way to calculate a + b.
 If the compiler doesn't recognize the simplification and optimize away all the unnecessary work, 
 this should create enough computational workload for the GPU to outperform the CPU.

 Write the loop as a for loop rather than a while loop. This will allow you to also use #pragma unroll 
 to explore whether it provides any speedup. Make sure to include an if (id < n) condition in your code 
 to ensure safety. Finally, be prepared to discuss the impact of #pragma unroll and whether it helped 
 improve performance.
*/

// Include files

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

// Defines
#define N 100000000 // Length of the vector                                                     //Change the length of the vector to be a very large number to test the GPU performace under
                                                                                                //a heavier load

// Global variables
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A_GPU, *B_GPU, *C_GPU; //GPU pointers
dim3 BlockSize; //This variable will hold the Dimensions of your blocks
dim3 GridSize; //This variable will hold the Dimensions of your grid
float Tolerance = 0.01;

// Function prototypes
void cudaErrorCheck(const char *, int);
void setUpDevices();
void allocateMemory();
void innitialize();
void addVectorsCPU(float*, float*, float*, int);
__global__ void addVectorsGPU(float, float, float, int);
bool  check(float*, int);
long elaspedTime(struct timeval, struct timeval);
void cleanUp();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

// This will be the layout of the parallel space we will be using.
void setUpDevices()
{
	BlockSize.x = 256;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	GridSize.x = 64; // This gives us the correct number of blocks.                     //Hard coded the GridSize.x to 64 like the instructions asked us to do 
	GridSize.y = 1;
	GridSize.z = 1;
}

// Allocating the memory we will be using.
void allocateMemory()
{	
	// Host "CPU" memory.				
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));
	
	// Device "GPU" Memory
	hipMalloc(&A_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
}

// Loading values into the vectors that we will add.
void innitialize()
{
	for(int i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)i;	
		B_CPU[i] = (float)(2*i);
	}
}

// Adding vectors a and b on the CPU then stores result in vector c.
void addVectorsCPU(float *a, float *b, float *c, int n)
{
	for(int id = 0; id < n; id++)
	{ 
		c[id] = sqrtf(cosf(a[id]) * cosf(a[id]) + a[id]*a[id] + sinf(a[id]) * sinf(a[id]) - 1.0f)      //Changed the a + b equation to this one to have more computation on the CPU
                                                                                                       //the way this adds more work is its doing more work on the kernel going back
                                                                                                       //and forth between the CPU and the GPU. This causes more time on the GPU and CPU 
                                                                                                       //doing math. This also helps reveal the advantages of parallel programming.
              + sqrtf(cosf(b[id])* cosf(b[id]) + b[id]*b[id] + sinf(b[id]) * sinf(b[id]) - 1.0f);
	}
}

// This is the kernel. It is the function that will run on the GPU.
// It adds vectors a and b on the GPU then stores result in vector c.
__global__ void addVectorsGPU(float *a, float *b, float *c, int n)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	#pragma unroll                                                                                      //This enables support for any vector size, and helps with performance tuning. 
                                                                                                        //This function expands the loop so that fewer iterations and steps are taken.
                                                                                                        //This boost the performance by elimination loop overhead and allows the GPU
                                                                                                        //to execute more instructions when unrolled thus speeding up the GPU a little 
                                                                                                        //and out performs the CPU.
	for (int i = id; i < n; i += blockDim.x * gridDim.x)
    {
        if (i < n)                                                                                      //This equation ensures that the CPU and GPU are doing the same work for the same reason
                                                                                                        //stated above.
        {
            c[i] = sqrtf(cosf(a[i]) * cosf(a[i]) + a[i]*a[i] + sinf(a[i]) * sinf(a[i]) - 1.0f)
                 + sqrtf(cosf(b[i]) * cosf(b[i]) + b[i]*b[i] + sinf(b[i]) * sinf(b[i]) - 1.0f);
        }
    }
}

// Checking to see if anything went wrong in the vector addition.
bool check(float *c, int n, float tolerence)
{
	int id;
	double myAnswer;
	double trueAnswer;
	double percentError;
	double m = n-1; // Needed the -1 because we start at 0.
	
	myAnswer = 0.0;
	for(id = 0; id < n; id++)
	{ 
		myAnswer += c[id];
	}
	
	trueAnswer = 3.0*(m*(m+1))/2.0;
	
	percentError = abs((myAnswer - trueAnswer)/trueAnswer)*100.0;
	
	if(percentError < Tolerance) 
	{
		return(true);
	}
	else 
	{
		return(false);
	}
}

// Calculating elasped time.
long elaspedTime(struct timeval start, struct timeval end)
{
	// tv_sec = number of seconds past the Unix epoch 01/01/1970
	// tv_usec = number of microseconds past the current second.
	
	long startTime = start.tv_sec * 1000000 + start.tv_usec; // In microseconds.
	long endTime = end.tv_sec * 1000000 + end.tv_usec; // In microseconds

	// Returning the total time elasped in microseconds
	return endTime - startTime;
}

// Cleaning up memory after we are finished.
void CleanUp()
{
	// Freeing host "CPU" memory.
	free(A_CPU); 
	free(B_CPU); 
	free(C_CPU);
	
	hipFree(A_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(B_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(C_GPU);
	cudaErrorCheck(__FILE__, __LINE__);
}

int main()
{
	timeval start, end;
	long timeCPU, timeGPU;
	
	// Setting up the GPU
	setUpDevices();
	
	// Allocating the memory you will need.
	allocateMemory();
	
	// Putting values in the vectors.
	innitialize();
	
	// Adding on the CPU
	gettimeofday(&start, NULL);
	addVectorsCPU(A_CPU, B_CPU ,C_CPU, N);
	gettimeofday(&end, NULL);
	timeCPU = elaspedTime(start, end);
	
	// Zeroing out the C_CPU vector just to be safe because right now it has the correct answer in it.
	for(int id = 0; id < N; id++)
	{ 
		C_CPU[id] = 0.0;
	}
	
	// Adding on the GPU
	gettimeofday(&start, NULL);
	
	// Copy Memory from CPU to GPU		
	hipMemcpyAsync(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	
	addVectorsGPU<<<GridSize,BlockSize>>>(A_GPU, B_GPU ,C_GPU, N);
	cudaErrorCheck(__FILE__, __LINE__);
	
	// Copy Memory from GPU to CPU	
	hipMemcpyAsync(C_CPU, C_GPU, N*sizeof(float), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	
	// Making sure the GPU and CPU wiat until each other are at the same place.
	hipDeviceSynchronize();
	cudaErrorCheck(__FILE__, __LINE__);
	
	gettimeofday(&end, NULL);
	timeGPU = elaspedTime(start, end);
	
	// Checking to see if all went correctly.
	if(check(C_CPU, N, Tolerance) == false)
	{
		printf("\n\n Something went wrong in the GPU vector addition\n");
	}
	else
	{
		printf("\n\n You added the two vectors correctly on the GPU");
		printf("\n The time it took on the CPU was %ld microseconds", timeCPU);
		printf("\n The time it took on the GPU was %ld microseconds", timeGPU);
	}
	
	// Your done so cleanup your room.	
	CleanUp();	
	
	// Making sure it flushes out anything in the print buffer.
	printf("\n\n");
	
	return(0);
}
