// Name:Tanner Wells 
// Page-locked memory test
// nvcc 13PageLockedMemory.cu -o temp

/*
 What to do:
 Read about **page-locked (pinned) memory**. Fill in the ???s in this code to understand how to
 set up and test page-locked memory on the host.
*/

/*
 Purpose:
 To learn how page-locked (pinned) memory works and how to use it effectively.
*/

// Include files
#include <sys/time.h>
#include <stdio.h>

// Defines
#define SIZE 2000000 
#define NUMBER_OF_COPIES 1000

//Globals
float *NumbersOnGPU, *PageableNumbersOnCPU, *PageLockedNumbersOnCPU;
hipEvent_t StartEvent, StopEvent;

//Function prototypes
void cudaErrorCheck(const char *, int);
void setUpCudaDevices();
void allocateMemory();
void cleanUp();
void copyPageableMemoryUp();
void copyPageLockedMemoryUp();
void copyPageableMemoryDown();
void copyPageLockedMemoryDown();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

//This will be the layout of the parallel space we will be using.
void setUpCudaDevices()
{
	hipEventCreate(&StartEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventCreate(&StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
}

//Sets a side memory on the GPU and CPU for our use.
void allocateMemory()
{					
	//Allocate Device (GPU) Memory
	hipMalloc(&NumbersOnGPU, SIZE*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);

	//Allocate pageable Host (CPU) Memory
	PageableNumbersOnCPU = (float*)malloc(SIZE*sizeof(float));
	
	//Allocate page locked Host (CPU) Memory
	hipHostMalloc(&PageLockedNumbersOnCPU, SIZE * sizeof(float));							//This allocates the page-locked memory 
	cudaErrorCheck(__FILE__, __LINE__);
}

//Cleaning up memory after we are finished.
void cleanUp()
{
	hipFree(NumbersOnGPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	
	hipHostFree(PageLockedNumbersOnCPU);													//this properly frees the page-locked memory when using page-locked memory
	cudaErrorCheck(__FILE__, __LINE__);
	
	free(PageableNumbersOnCPU); 
	
	hipEventDestroy(StartEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventDestroy(StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
}

void copyPageableMemoryUp()
{
	for(int i = 0; i < NUMBER_OF_COPIES; i++)
	{
		hipMemcpy(NumbersOnGPU, PageableNumbersOnCPU, SIZE*sizeof(float), hipMemcpyHostToDevice);
		cudaErrorCheck(__FILE__, __LINE__);
	}
}

void copyPageableMemoryDown()
{
	for(int i = 0; i < NUMBER_OF_COPIES; i++)
	{
		hipMemcpy(PageableNumbersOnCPU, NumbersOnGPU, SIZE*sizeof(float), hipMemcpyDeviceToHost);					//this uses the same function doesnt change but the inside changes as the CUDA skips the internal step
		cudaErrorCheck(__FILE__, __LINE__);
	}
}

void copyPageLockedMemoryUp()
{
	for(int i = 0; i < NUMBER_OF_COPIES; i++)
	{
		hipMemcpy(NumbersOnGPU, PageLockedNumbersOnCPU, SIZE * sizeof(float), hipMemcpyHostToDevice);
		cudaErrorCheck(__FILE__, __LINE__);
	}
}

void copyPageLockedMemoryDown()
{
	for(int i = 0; i < NUMBER_OF_COPIES; i++)
	{
		hipMemcpy(PageLockedNumbersOnCPU, NumbersOnGPU, SIZE * sizeof(float), hipMemcpyDeviceToHost);				//This is does the same as the function call above 
		cudaErrorCheck(__FILE__, __LINE__);
	}
}

int main()
{
	float timeEvent;
	
	setUpCudaDevices();
	allocateMemory();
	
	hipEventRecord(StartEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	copyPageableMemoryUp();
	hipEventRecord(StopEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventSynchronize(StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU using pageable memory up = %3.1f milliseconds", timeEvent);
	
	hipEventRecord(StartEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	copyPageLockedMemoryUp();
	hipEventRecord(StopEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventSynchronize(StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU using page locked memory up = %3.1f milliseconds", timeEvent);
	
	hipEventRecord(StartEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	copyPageableMemoryDown();
	hipEventRecord(StopEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventSynchronize(StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU using pageable memory down = %3.1f milliseconds", timeEvent);
	
	hipEventRecord(StartEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	copyPageLockedMemoryDown();
	hipEventRecord(StopEvent, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	hipEventSynchronize(StopEvent);
	cudaErrorChecmyCudaErrorCheckk(__FILE__, __LINE__);
	hipEventElapsedTime(&timeEvent, StartEvent, StopEvent);
	cudaErrorCheck(__FILE__, __LINE__);
	printf("\n Time on GPU using page locked memory down = %3.1f milliseconds", timeEvent);
	
	printf("\n");
	//You're done so cleanup your mess.
	cleanUp();	
	
	return(0);
}
